#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <string>
#include <cmath>
#include <iomanip>
#include <fstream>
#include <algorithm>
#include <bits/stdc++.h>
#include <omp.h>
#include "mpi.h"
using namespace std;
// Индексация внутри блока
#define _i(i, j) (((j) + 1) * (x_b + 2) + (i) + 1)
#define _ix(id) (((id) % (x_b + 2)) - 1)
#define _iy(id) (((id) / (y_b + 2)) - 1)
// Индексация по блокам (процессам)
#define _ib(i, j) ((j) * x_g + (i))
#define _ibx(id) ((id) % x_g)
#define _iby(id) ((id) / x_g)

typedef unsigned uint;
typedef double dbl;

int main(int argc, char* argv[]) {
	ios_base::sync_with_stdio(false);
	cin.tie(NULL);
	cout.tie(NULL);

	dbl 
		face_u, 
		tail_u, 
		left_u, 
		right_u,
		first_u,
		lx, ly,
		eps;

	uint x_g, y_g, x_b, y_b;

	string out_file;

	int 
		id, ib, jb,
		i, j,
		count;
	dbl * data, * temp, * next, * buff,
		* x_s, * y_s;

	MPI_Status status;
	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &count);
	MPI_Comm_rank(MPI_COMM_WORLD, &id);

	MPI_Barrier(MPI_COMM_WORLD);

	if (id < 1) {
		std::cin >> x_g >> y_g;
		std::cin >> x_b >> y_b;
		std::cin >> out_file;
		std::cin >> eps;
		std::cin >> lx >> ly;
		std::cin >> left_u >> right_u >> face_u >> tail_u;
		std::cin >> first_u;
	}
	// Передача параметров расчета всем процессам
	MPI_Bcast(&x_b, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&y_b, 1, MPI_INT, 0, MPI_COMM_WORLD);

	MPI_Bcast(&x_g, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&y_g, 1, MPI_INT, 0, MPI_COMM_WORLD);

	MPI_Bcast(&lx, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&ly, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

	MPI_Bcast(&eps, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

	MPI_Bcast(&tail_u, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&face_u, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&left_u, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&right_u, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&first_u, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

	ib = _ibx(id);		// Переход к 2-мерной индексации процессов 
	jb = _iby(id);

	dbl 
		hx = lx / (dbl)(x_g * x_b),
		hy = ly / (dbl)(y_g * y_b),
		sz_max,
		err_max;

	data = (dbl*)malloc(sizeof(dbl) * (x_b + 2) * (y_b + 2));
	next = (dbl*)malloc(sizeof(dbl) * (x_b + 2) * (y_b + 2));

	sz_max = max(x_b, y_b);

	x_s = (dbl*)malloc(sizeof(dbl) * sz_max);
	y_s = (dbl*)malloc(sizeof(dbl) * sz_max);

	buff = (dbl*)malloc(sizeof(dbl) * sz_max);

	for (i = 0; i < x_b; i++)					// Инициализация блока
		for (j = 0; j < y_b; j++)
			data[_i(i, j)] = first_u;

	err_max = 0.0;
	omp_set_num_threads(2);
	do {
		MPI_Barrier(MPI_COMM_WORLD);
		// Отправка данных
		if (ib < x_g - 1) {
			j = 0;
			while(j < y_b){
				y_s[j] = data[_i(x_b - 1, j)];
				j++;
			}	
			MPI_Bsend(y_s, y_b, MPI_DOUBLE, _ib(ib + 1, jb), 0, MPI_COMM_WORLD);
		}

		if (jb < y_g - 1) {
			i = 0;
			while(i < x_b){
				x_s[i] = data[_i(i, y_b - 1)];
				i++;
			}	
			MPI_Bsend(x_s, x_b, MPI_DOUBLE, _ib(ib, jb + 1), 0, MPI_COMM_WORLD);
		}
		// Прием данных
		if (ib <= 0) {
			#pragma omp parallel for private(i, j) shared(data)
			for(j = 0; j < y_b; ++j){
				data[_i(-1, j)] = left_u;	
			}	
		}
		else {
			MPI_Recv(y_s, y_b, MPI_DOUBLE, _ib(ib - 1, jb), 0, MPI_COMM_WORLD, &status);
			#pragma omp parallel for private(i, j) shared(data, y_s)
			for(j = 0; j < y_b; ++j){
				data[_i(-1, j)] = y_s[j];
			}	
		}

		if (jb <= 0) {
			#pragma omp parallel for private(i, j) shared(data)
			for(i = 0; i < x_b; ++i){
				data[_i(i, -1)] = face_u;
			}	
		}
		else {
			MPI_Recv(x_s, x_b, MPI_DOUBLE, _ib(ib, jb - 1), 0, MPI_COMM_WORLD, &status);
			#pragma omp parallel for private(i, j) shared(data, x_s)
			for (i = 0; i < x_b; ++i){
				data[_i(i, -1)] = x_s[i];
			}
		}

		//Отправить
		if (ib >= 1) {
			j = 0;
			while(j < y_b){
				y_s[j] = data[_i(0, j)];
				j++;
			}
			int tempBlockSize = y_b;
			MPI_Bsend(y_s, tempBlockSize, MPI_DOUBLE, _ib(ib - 1, jb), 0, MPI_COMM_WORLD);
		}

		if (jb >= 1) {
			i = 0;
			while(i < x_b){
				x_s[i] = data[_i(i, 0)];
				i++;
			}	
			int tempBlockSize = x_b;
			MPI_Bsend(x_s, tempBlockSize, MPI_DOUBLE, _ib(ib, jb - 1), 0, MPI_COMM_WORLD);
		}

		//Принять
		if (ib >= x_g - 1) {
			#pragma omp parallel for private(i, j) shared(data)
			for (j = 0; j < y_b; j++){
				data[_i(x_b, j)] = right_u;				
			}
		}
		else {
			int tempBlockSize = y_b;
			MPI_Recv(y_s, tempBlockSize, MPI_DOUBLE, _ib(ib + 1, jb), 0, MPI_COMM_WORLD, &status);
			#pragma omp parallel for private(i, j) shared(data, y_s)
			for (j = 0; j < y_b; j++){
				data[_i(x_b, j)] = y_s[j];
			}	
		}

		if (jb >= y_g - 1) {
			i = 0;
			while(i < x_b){
				data[_i(i, y_b)] = tail_u;
				i++;
			}		
		}
		else {
			int tempBlockSize = x_b;
			MPI_Recv(x_s, tempBlockSize, MPI_DOUBLE, _ib(ib, jb + 1), 0, MPI_COMM_WORLD, &status);
			#pragma omp parallel for private(i, j) shared(data, x_s)
			for (i = 0; i < x_b; i++){
				data[_i(i, y_b)] = x_s[i];
			}	
		}

		MPI_Barrier(MPI_COMM_WORLD);

		//		Перевычисление значений температуры
		#pragma omp parallel for private(i, j) shared(data, next)
		for (i = 0; i < x_b; i++) {
			for (j = 0; j < y_b; j++) {
				next[_i(i, j)] = 0.5 * ((data[_i(i + 1, j)] + data[_i(i - 1, j)]) / (hx * hx) +
					(data[_i(i, j + 1)] + data[_i(i, j - 1)]) / (hy * hy)) /
					(1.0 / (hx * hx) + 1.0 / (hy * hy));
				err_max = max(err_max, abs(next[_i(i, j)] - data[_i(i, j)]));
			}
		}
		//error
		err_max = 0.0;
		#pragma omp parallel for private(i, j) shared(data, next) reduction(max: err_max)
		for (i = 0; i < x_b; ++i) {
			for (j = 0; j < y_b; ++j) {
				err_max = max(err_max, abs(next[_i(i, j)] - data[_i(i, j)]));
			}
		}
		MPI_Allreduce(&err_max, &err_max, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

		temp = next;
		next = data;
		data = temp;
	} while (err_max >= eps);

	MPI_Barrier(MPI_COMM_WORLD);

	if (id == 0) {
		std::fstream file(out_file, std::ios::out);
		file << std::scientific << std::setprecision(6);

		for (jb = 0; jb < y_g; jb++)
			for (j = 0; j < y_b; j++)
				for (ib = 0; ib < x_g; ib++) {
					if (_ib(ib, jb) != 0) {
						int tempBlockSize = x_b;
						MPI_Recv(buff, tempBlockSize, MPI_DOUBLE, _ib(ib, jb), 0, MPI_COMM_WORLD, &status);
					}
					else {
						i = 0;
						while(i < x_b){
							buff[i] = data[_i(i, j)];
							i++;
						}
					}
					i = 0;
					while(i < x_b){
						fprintf(stderr, "%.6e ", buff[i]);
						file << buff[i] << " ";
						i++;
					}
				}	
	}
	else {
		for (j = 0; j < y_b; j++) {
			for (i = 0; i < x_b; i++)
				buff[i] = data[_i(i, j)];
			int tempBlockSize = x_b;
			MPI_Bsend(buff, tempBlockSize, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD);
		}
	}
	MPI_Barrier(MPI_COMM_WORLD);
	MPI_Finalize();

	free(buff);
	free(data);
	free(next);
	return 0;
}