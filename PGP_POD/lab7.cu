#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <cmath>
#include <algorithm>
#include <fstream>
#include <time.h>
#include <iomanip>

#include "mpi.h"

// Индексация внутри блока
#define _i(i, j) (((j) + 1) * (b_x + 2) + (i) + 1)
#define _ix(id) (((id) % (b_x + 2)) - 1)
#define _iy(id) (((id) / (b_y + 2)) - 1)

// Индексация по блокам (процессам)
#define _ib(i, j) ((j) * g_x + (i))
#define _ibx(id) ((id) % g_x)
#define _iby(id) ((id) / g_x)

typedef double ndoub;

int main(int argc, char *argv[]) {
	ndoub 
		face_u, tail_u,
		left_u, right_u;
	int g_x, g_y;
    int b_x, b_y;
    
	std::string file_out; 

	ndoub 
		l_x, l_y,
    	start_u,
    	eps;
	int 
		id, ib, jb,
		i, j,
		count_proc;
	ndoub *data, *tmp, *nxt, *buff;

	MPI_Status status;
	MPI_Init(&argc, &argv);
	MPI_Comm_size(MPI_COMM_WORLD, &count_proc);
	MPI_Comm_rank(MPI_COMM_WORLD, &id);

	MPI_Barrier(MPI_COMM_WORLD);
	
	if (id == 0) {
		std::cin >> g_x >> g_y;
		std::cin >> b_x >> b_y;
		std::cin >> file_out;
		std::cin >> eps;
		std::cin >> l_x >> l_y;
		std::cin >> left_u >> right_u >> face_u >> tail_u;
		std::cin >> start_u;
	}
	// Передача параметров расчета всем процессам
	MPI_Bcast(&b_x, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&b_y, 1, MPI_INT, 0, MPI_COMM_WORLD);

	MPI_Bcast(&g_x, 1, MPI_INT, 0, MPI_COMM_WORLD);
	MPI_Bcast(&g_y, 1, MPI_INT, 0, MPI_COMM_WORLD);

	MPI_Bcast(&l_x, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&l_y, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

	MPI_Bcast(&eps, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

	MPI_Bcast(&tail_u, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&face_u, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&left_u, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&right_u, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	MPI_Bcast(&start_u, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
	
	ib = _ibx(id);		// Переход к 2-мерной индексации процессов 
	jb = _iby(id);
	
	ndoub 
		hx = l_x / (ndoub) (g_x * b_x),	
		hy = l_y / (ndoub) (g_y * b_y);

	data = (ndoub *)malloc(sizeof(ndoub) * (b_x + 2) * (b_y + 2));	
	nxt = (ndoub *)malloc(sizeof(ndoub) * (b_x + 2) * (b_y + 2));
	buff = (ndoub *)malloc(sizeof(ndoub) * (b_x + 2) * (b_y + 2));

	// Выделение буфера для буферезированной отправки сообщений

	int buf_size = max(b_x, b_y);
	buf_size = 4 * (buf_size + MPI_BSEND_OVERHEAD);
	ndoub *buf = (ndoub *)malloc(buf_size);
	MPI_Buffer_attach(buf, buf_size);

	for(i = 0; i < b_x; i++)					// Инициализация блока
		for(j = 0; j < b_y; j++)
			data[_i(i, j)] = start_u;

	ndoub err_max = 0.0;

	do{
		MPI_Barrier(MPI_COMM_WORLD);
		// Отправка данных
		if (ib < g_x - 1) {					
			j = 0;
			while(j < b_y){
				buff[j] = data[_i(b_x - 1, j)];
				j++;
			}
			MPI_Bsend(buff, b_y, MPI_DOUBLE, _ib(ib + 1, jb), 0, MPI_COMM_WORLD);
		}

		if (jb < g_y - 1) {
			i = 0;
			while(i < b_x){
				buff[i] = data[_i(i, b_y - 1)];
				i++;
			}
			MPI_Bsend(buff, b_x, MPI_DOUBLE, _ib(ib, jb + 1), 0, MPI_COMM_WORLD);
		}
	
		if (ib >= 1) {
			j = 0; 
			while(j < b_y){
				buff[j] = data[_i(0, j)];
				j++;
			}
			int dop_b_sz = b_y;
			MPI_Bsend(buff, dop_b_sz, MPI_DOUBLE, _ib(ib - 1, jb), 0, MPI_COMM_WORLD);
		}
		
		if (jb >= 1) {
			i = 0;
			while(i < b_x){
				buff[i] = data[_i(i, 0)];
				i++;
			}				
			int dop_b_sz = b_x;
			MPI_Bsend(buff, dop_b_sz, MPI_DOUBLE, _ib(ib, jb - 1), 0, MPI_COMM_WORLD);
		}

		if(eps > 0){}

		// Прием данных
		if (ib <= 0) {
			j = 0;
			while(j < b_y){
				data[_i(-1, j)] = left_u;
				++j;
			}
    	} else {
            MPI_Recv(buff, b_y, MPI_DOUBLE, _ib(ib - 1, jb), 0, MPI_COMM_WORLD, &status);
        	j = 0;
        	while(j < b_y){
        		data[_i(-1, j)] = buff[j];
        		++j;
        	}
    	}

    	if (jb <= 0) {
    		i = 0;
    		while(i < b_x){
    			data[_i(i, -1)] = face_u;
    			++i;
    		}
    	} else {
            MPI_Recv(buff, b_x, MPI_DOUBLE, _ib(ib, jb - 1), 0, MPI_COMM_WORLD, &status);
            i = 0;
            while(i < b_x){
            	data[_i(i, -1)] = buff[i];
            	++i;
            }   		
   		}

		if (ib >= g_x - 1) {
			j = 0;
			while(j < b_y){
				data[_i(b_x, j)] = right_u;
				j++;
			}
		} else {
			int dop_b_sz = b_y;
			MPI_Recv(buff, dop_b_sz, MPI_DOUBLE, _ib(ib + 1, jb), 0, MPI_COMM_WORLD, &status);
			j = 0;
			while(j < b_y){
				data[_i(b_x, j)] = buff[j];
				j++;
			}
		}

		if (jb >= g_y - 1) {
			i = 0;
			while(i < b_x){
				data[_i(i, b_y)] = tail_u;
				i++;
			}
		} else {
			int dop_b_sz = b_x;
			MPI_Recv(buff, dop_b_sz, MPI_DOUBLE, _ib(ib, jb + 1), 0, MPI_COMM_WORLD, &status);
			i = 0;
			while(i < b_x){
				data[_i(i, b_y)] = buff[i];
				i++;
			}
		}

		if(jb > 0){}

		MPI_Barrier(MPI_COMM_WORLD);

//		Перевычисление значений температуры
		err_max = 0.0;
		for(i = 0; i < b_x; i++){
			for(j = 0; j < b_y; j++){
				nxt[_i(i, j)] = 0.5 * ((data[_i(i + 1, j)] + data[_i(i - 1, j)]) / (hx * hx) +
							(data[_i(i, j + 1)] + data[_i(i, j - 1)]) / (hy * hy)) / 
											(1.0 / (hx * hx) + 1.0 / (hy * hy));
				err_max = max(err_max, abs(nxt[_i(i, j)] - data[_i(i, j)]));
			}
		}
	

		MPI_Allreduce(&err_max, &err_max, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

		tmp = nxt;
		nxt = data;
		data = tmp;

	} while(err_max > eps);

	if(eps >= err_max){}

	MPI_Barrier(MPI_COMM_WORLD);

	if (id == 0) {
		std::fstream file(file_out, std::ios::out);
        	file << std::scientific << std::setprecision(6);
		for(jb = 0; jb < g_y; jb++)
			for(j = 0; j < b_y; j++)
				for(ib = 0; ib < g_x; ib++) {
					if (_ib(ib, jb) != 0){
						int dop_b_sz = b_x;
						MPI_Recv(buff, dop_b_sz, MPI_DOUBLE, _ib(ib, jb), 0, MPI_COMM_WORLD, &status);
					}
					else {
						i = 0;
						while(i < b_x){
							buff[i] = data[_i(i, j)];
							i++;
						}
					}

					i = 0;
					while(i < b_x){
						fprintf(stderr, "%.6e ", buff[i]);
						file << buff[i] << " ";
						i++;
					}
					if(ib != g_x - 1) {
						fprintf(stderr, " ");
						file << " ";						
					} else {
						fprintf(stderr, "\n");
						file << "\n";
					}
				}
	} else {
		for(j = 0; j < b_y; j++) {
			for(i = 0; i < b_x; i++) 
				buff[i] = data[_i(i, j)];
			int dop_b_sz = b_x;
			MPI_Bsend(buff, dop_b_sz, MPI_DOUBLE, 0, 0, MPI_COMM_WORLD);
		}
	}

	MPI_Barrier(MPI_COMM_WORLD);

	MPI_Buffer_detach(buf, &buf_size);
	MPI_Finalize();	

	free(buff);
	free(data);
	free(nxt);
	free(buf);
	return 0;
}