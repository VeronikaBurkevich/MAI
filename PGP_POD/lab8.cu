#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <mpi.h>
#include <omp.h>

typedef unsigned uint;
typedef double dbl;

#include <thrust/extrema.h>
            #include <thrust/device_vector.h>

            #define CSC(call)                                            \
            do {                                                         \
        hipError_t res = call;                                          \
        if (res != hipSuccess) {                                        \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n",             \
                    __FILE__, __LINE__, hipGetErrorString(res));        \
            exit(0);                                                     \
        }                                                                \
    } while(0)

            #define _i(i, j) (((j) + 1) * (x_block + 2) + (i) + 1)
            #define _ix(id) (((id) % (x_block + 2)) - 1)
            #define _iy(id) (((id) / (y_block + 2)) - 1)

            #define _ib(i, j) ((j) * x_gr + (i))
            #define _ibx(id) ((id) % x_gr)
            #define _iby(id) ((id) / x_gr)

    __global__ static void kernel_copy_x(
        dbl *side, 
        dbl *data, 
        int x_block, 
        int j, 
        bool direction, 
        dbl value
    ) {
        const uint 
            id = blockIdx.x * blockDim.x + threadIdx.x,
            offset = blockDim.x * gridDim.x;
        int i;

        if (!direction) {
            if (!side) {
                i = id;
                while(i < x_block){
                    data[_i(i, j)] = value;
                    i += offset;
                }
            } else{
                i = id;
                while(i < x_block){
                    data[_i(i, j)] = side[i];
                    i += offset;
                }
            }
        } else {
            i = id;
            while(i < x_block){
                side[i] = data[_i(i, j)];
                i += offset;
            }
        }
    }

    __global__ static void kernel_copy_y(
        dbl *side, 
        dbl *data, 
        int y_block,
        int x_block,  
        int i, 
        bool direction, 
        dbl value
    ) {
        const uint 
            id = blockIdx.y * blockDim.y + threadIdx.y,
            offset = blockDim.y * gridDim.y;
        int j;

        if (!direction) {
            if (!side) {
                j = id;
                while(j < y_block){
                    data[_i(i, j)] = value;
                    j += offset;
                }            
            } else{
                j = id;
                while(j < y_block){
                    data[_i(i, j)] = side[j];
                    j += offset;
                }
            }
        } else {
            j = id;
            while(j < y_block){
                side[j] = data[_i(i, j)];
                j += offset;
            }
        }
    }

    __global__ static void kernel_calculate_temp(
        dbl* next, 
        dbl* data, 
        int x_block, 
        int y_block, 
        dbl hx, 
        dbl hy
    ){
        const uint 
            x_id = blockIdx.x * blockDim.x + threadIdx.x,
            y_id = blockIdx.y * blockDim.y + threadIdx.y,
            x_offset = blockDim.x * gridDim.x,
            y_offset = blockDim.y * gridDim.y;
        int i, j;

        for (i = x_id; i < x_block; i += x_offset) {
            for (j = y_id; j < y_block; j += y_offset) {
                dbl 
                    u_i = (data[_i(i + 1, j)] + data[_i(i - 1, j)]) / hx,
                    u_j = (data[_i(i, j + 1)] + data[_i(i, j - 1)]) / hy;

                next[_i(i, j)] = 0.5 * (u_i + u_j) / (1.0 / hx + 1.0 / hy);
            }
        }
    }

    __global__ static void kernel_errors(
        dbl* next, 
        dbl* data, 
        dbl* diff, 
        int x_block, 
        int y_block
    ){
        const uint 
            x_id = blockIdx.x * blockDim.x + threadIdx.x,
            y_id = blockIdx.y * blockDim.y + threadIdx.y,
            x_offset = blockDim.x * gridDim.x,
            y_offset = blockDim.y * gridDim.y;
        int i, j;

        for (i = x_id - 1; i <= x_block; i += x_offset) {
            for (j = y_id - 1; j <= y_block; j += y_offset) {
                diff[_i(i, j)] = (i != -1 && j != -1 && i != x_block && j != y_block) ? abs(next[_i(i, j)] - data[_i(i, j)]) : 0;
            }
        }

    }

    int main(int argc, char *argv[]) {
        int id, 
            x_block, y_block, 
            x_gr, y_gr;
        int i, j;
        int count, outputFileLength = 256;
        char out_file[256];
        dbl lx, ly, 
            backU, frontU, 
            leftU, rightU, 
            startU,
            eps, 
            maxError = 0.0;

        MPI_Status status;
        MPI_Init(&argc, &argv);
        MPI_Comm_size(MPI_COMM_WORLD, &count);
        MPI_Comm_rank(MPI_COMM_WORLD, &id);

        fprintf(stderr, "proc %d(%d)\n", id, count);
        fflush(stderr);

        MPI_Barrier(MPI_COMM_WORLD);

        if (id < 1) {
            std::cin >> x_gr >> y_gr;
            std::cin >> x_block >> y_block;
            std::cin >> out_file;
            std::cin >> eps;
            std::cin >> lx >> ly;
            std::cin >> leftU >> rightU >> frontU >> backU;
            std::cin >> startU;
        }

        MPI_Bcast(&x_gr, 1, MPI_INT, 0, MPI_COMM_WORLD);
        MPI_Bcast(&y_gr, 1, MPI_INT, 0, MPI_COMM_WORLD);

        MPI_Bcast(&x_block, 1, MPI_INT, 0, MPI_COMM_WORLD);
        MPI_Bcast(&y_block, 1, MPI_INT, 0, MPI_COMM_WORLD);

        MPI_Bcast(&eps, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

        MPI_Bcast(&lx, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
        MPI_Bcast(&ly, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

        MPI_Bcast(&backU, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
        MPI_Bcast(&frontU, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
        MPI_Bcast(&leftU, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);
        MPI_Bcast(&rightU, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

        MPI_Bcast(&startU, 1, MPI_DOUBLE, 0, MPI_COMM_WORLD);

        outputFileLength = strlen(out_file) + 1;
        MPI_Bcast(&outputFileLength, 1, MPI_INT, 0, MPI_COMM_WORLD);
        MPI_Bcast(out_file, outputFileLength, MPI_CHAR, 0, MPI_COMM_WORLD);

        if (id < 1) {
            fprintf(stderr, "x_gr: %d y_gr: %d\n", x_gr, y_gr);
            fprintf(stderr, "x_block: %d y_block: %d\n", x_block, y_block);
            fprintf(stderr, "eps: %lf\n", eps);
            fprintf(stderr, "lx: %lf ly: %lf\n", lx, ly);
            fprintf(stderr, "backU: %lf frontU: %lf leftU: %lf rightU: %lf\n", backU, frontU, leftU, rightU);
            fprintf(stderr, "startU: %lf\n", startU);
        }

        if (x_gr * y_gr * x_block * y_block == 0) {
            fprintf(stderr, "Error at proc %d\n", id);
            if (x_gr * y_gr != count) {
                fprintf(stderr, "Dead because of blocks\n");
                fprintf(stderr, "x_gr=%d, y_gr=%d, count=%d\n", x_gr, y_gr, count);
            }
            fflush(stderr);
            MPI_Finalize();
            return 0;
        }

        int 
            ib = _ibx(id),
            jb = _iby(id);

        dbl 
            hx = lx / ((dbl) x_gr * (dbl) x_block),
            hy = ly / ((dbl) y_gr * (dbl) y_block);

        hx *= hx;
        hy *= hy;

        dbl size = (x_block + 2) * (y_block + 2);
 
        dbl
            * data = (dbl *)malloc(sizeof(dbl) * size),
            * next = (dbl *)malloc(sizeof(dbl) * size),
            * edge_x = (dbl *)malloc(sizeof(dbl) * x_block),
            * edge_y = (dbl *)malloc(sizeof(dbl) * y_block);

        for (i = 0; i < x_block; i++) {
            for (j = 0; j < y_block; j++) {
                data[_i(i, j)] = startU;
            }
        }

        dbl             //
            *gpu_data,     //
            *gpu_next,     //
            *gpu_error,    //
            *gpu_edge_x,   //
            *gpu_edge_y;   //

        CSC(hipMalloc(&gpu_data, sizeof(dbl) * size));
        CSC(hipMemcpy(gpu_data, data, sizeof(dbl) * size, hipMemcpyHostToDevice));

        CSC(hipMalloc(&gpu_next, sizeof(dbl) * size));
        CSC(hipMemcpy(gpu_next, next, sizeof(dbl) * size, hipMemcpyHostToDevice));

        CSC(hipMalloc(&gpu_error, sizeof(dbl) * size));

        CSC(hipMalloc(&gpu_edge_x, sizeof(dbl) * x_block));
        CSC(hipMemcpy(gpu_edge_x, edge_x, sizeof(dbl) * x_block, hipMemcpyHostToDevice));

        CSC(hipMalloc(&gpu_edge_y, sizeof(dbl) * y_block));
        CSC(hipMemcpy(gpu_edge_y, edge_y, sizeof(dbl) * y_block, hipMemcpyHostToDevice));

        dim3 blocks(32, 32);
        dim3 threads(32, 32);

        do {
            if (ib < x_gr - 1) {
                kernel_copy_y<<<blocks, threads>>>(gpu_edge_y, gpu_data, y_block, x_block, x_block - 1, true, startU);
                CSC(hipGetLastError());
                CSC(hipMemcpy(edge_y, gpu_edge_y, sizeof(dbl) * y_block, hipMemcpyDeviceToHost));
                MPI_Bsend(edge_y, y_block, MPI_DOUBLE, _ib(ib + 1, jb), 0, MPI_COMM_WORLD);
            }
            if (jb < y_gr - 1) {
                kernel_copy_x<<<blocks, threads>>>(gpu_edge_x, gpu_data, x_block, y_block - 1, true, startU);
                CSC(hipGetLastError());
                CSC(hipMemcpy(edge_x, gpu_edge_x, sizeof(dbl) * x_block, hipMemcpyDeviceToHost));
                MPI_Bsend(edge_x, x_block, MPI_DOUBLE, _ib(ib, jb + 1), 0, MPI_COMM_WORLD);
            }

            if (ib <= 0) {
                kernel_copy_y<<<blocks, threads>>>(NULL, gpu_data, y_block, x_block, -1, false, leftU);
                CSC(hipGetLastError());
            } else {
                MPI_Recv(edge_y, y_block, MPI_DOUBLE, _ib(ib - 1, jb), 0, MPI_COMM_WORLD, &status);
                CSC(hipMemcpy(gpu_edge_y, edge_y, sizeof(dbl) * y_block, hipMemcpyHostToDevice));
                kernel_copy_y<<<blocks, threads>>>(gpu_edge_y, gpu_data, y_block, x_block, -1, false, startU);
                CSC(hipGetLastError());
            }

            if (jb <= 0) {
                kernel_copy_x<<<blocks, threads>>>(NULL, gpu_data, x_block, -1, false, frontU);
                CSC(hipGetLastError());
            } else {
                MPI_Recv(edge_x, x_block, MPI_DOUBLE, _ib(ib, jb - 1), 0, MPI_COMM_WORLD, &status);
                CSC(hipMemcpy(gpu_edge_x, edge_x, sizeof(dbl) * x_block, hipMemcpyHostToDevice));
                kernel_copy_x<<<blocks, threads>>>(gpu_edge_x, gpu_data, x_block, -1, false, startU);
                CSC(hipGetLastError());
            }

            if (ib >= 1) {
                kernel_copy_y<<<blocks, threads>>>(gpu_edge_y, gpu_data, y_block, x_block, 0, true, startU);
                CSC(hipGetLastError());
                CSC(hipMemcpy(edge_y, gpu_edge_y, sizeof(dbl) * y_block, hipMemcpyDeviceToHost));
                MPI_Bsend(edge_y, y_block, MPI_DOUBLE, _ib(ib - 1, jb), 0, MPI_COMM_WORLD);
            }

            if (jb >= 1) {
                kernel_copy_x<<<blocks, threads>>>(gpu_edge_x, gpu_data, x_block, 0, true, startU);
                CSC(hipGetLastError());
                CSC(hipMemcpy(edge_x, gpu_edge_x, sizeof(dbl) * x_block, hipMemcpyDeviceToHost));
                MPI_Bsend(edge_x, x_block, MPI_DOUBLE, _ib(ib, jb - 1), 0, MPI_COMM_WORLD);
            }

            if (ib >= x_gr - 1) {
                kernel_copy_y<<<blocks, threads>>>(NULL, gpu_data, y_block, x_block, x_block, false, rightU);
                CSC(hipGetLastError());
            } else {
                MPI_Recv(edge_y, y_block, MPI_DOUBLE, _ib(ib + 1, jb), 0, MPI_COMM_WORLD, &status);
                hipMemcpy(gpu_edge_y, edge_y, sizeof(dbl) * y_block, hipMemcpyHostToDevice);
                kernel_copy_y<<<blocks, threads>>>(gpu_edge_y, gpu_data, y_block, x_block, x_block, false, startU);
                CSC(hipGetLastError());
            }

            if (jb >= y_gr - 1) {
                kernel_copy_x<<<blocks, threads>>>(NULL, gpu_data, x_block, y_block, false, backU);
                CSC(hipGetLastError());
            } else {
                MPI_Recv(edge_x, x_block, MPI_DOUBLE, _ib(ib, jb + 1), 0, MPI_COMM_WORLD, &status);
                hipMemcpy(gpu_edge_x, edge_x, sizeof(dbl) * x_block, hipMemcpyHostToDevice);
                kernel_copy_x<<<blocks, threads>>>(gpu_edge_x, gpu_data, x_block, y_block, false, startU);
                CSC(hipGetLastError());
            }

            hipDeviceSynchronize();
            kernel_calculate_temp<<<dim3(8, 8, 8), dim3(32, 4, 4)>>>(gpu_next, gpu_data, x_block, y_block, hx, hy);
            CSC(hipGetLastError());

            hipDeviceSynchronize();
            kernel_errors<<<dim3(8, 8, 8), dim3(32, 4, 4)>>>(gpu_next, gpu_data, gpu_error, x_block, y_block);
            CSC(hipGetLastError());

            thrust::device_ptr<dbl> p_arr = thrust::device_pointer_cast(gpu_error);
            thrust::device_ptr<dbl> thrust_max = thrust::max_element(p_arr, p_arr + size);

            dbl gpu_err = 0.0;
            gpu_err = *thrust_max;

            MPI_Allreduce(&gpu_err, &maxError, 1, MPI_DOUBLE, MPI_MAX, MPI_COMM_WORLD);

            dbl *temp = gpu_data;
            gpu_data = gpu_next;
            gpu_next = temp;

        } while (maxError > eps);

        MPI_Barrier(MPI_COMM_WORLD);

        CSC(hipMemcpy(data, gpu_data, sizeof(dbl) * size, hipMemcpyDeviceToHost));
        CSC(hipFree(gpu_data));
        CSC(hipFree(gpu_next));
        CSC(hipFree(gpu_error));
        CSC(hipFree(gpu_edge_x));
        CSC(hipFree(gpu_edge_y));

        int 
            nSize = 14,
            b_size = x_block * y_block,
            nBlockSize = b_size * nSize;

        char* buff = (char *)malloc(sizeof(char) * nBlockSize);
        memset(buff, ' ', nBlockSize * sizeof(char));

        for(j = 0; j < y_block; j++) {
            for (i = 0; i < x_block; i++) {
                sprintf(buff + (j * x_block + i) * nSize, " %.6e", data[_i(i, j)]);
            }
            if (ib == x_gr - 1) {
                buff[(j + 1) * x_block * nSize - 1] = '\n';
            }
        }

        i = 0;
        while (i < nBlockSize){
            if (buff[i] == '\0') {
                buff[i] = ' ';
            }
            ++i;
        }

        MPI_Datatype numType, rowType, filetype;

        MPI_Type_contiguous(nSize, MPI_CHAR, &numType);
        MPI_Type_contiguous(x_block, numType, &rowType);

        MPI_Type_create_hvector(y_block, 1, x_block * x_gr * nSize, rowType, &filetype);
        MPI_Type_commit(&filetype);

        MPI_File fp;
        MPI_File_delete(out_file, MPI_INFO_NULL);
        MPI_File_open(MPI_COMM_WORLD, out_file, MPI_MODE_CREATE | MPI_MODE_WRONLY, MPI_INFO_NULL, &fp);

        MPI_File_set_view(fp, (jb * b_size * x_gr + ib * x_block) * nSize, MPI_CHAR, filetype, "native", MPI_INFO_NULL);
        MPI_File_write_all(fp, buff, nBlockSize, MPI_CHAR, MPI_STATUS_IGNORE);

        MPI_File_close(&fp);

        MPI_Finalize();

        free(buff);
        free(data);
        free(next);
        free(edge_x);
        free(edge_y);

        return 0;
}