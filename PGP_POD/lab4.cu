#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <fstream>
#include <iomanip>
#include <iostream>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

using namespace std;

typedef unsigned uint;
typedef double dbl;

#define TITLE_DIM (32U)

#define exit_if(cnd_value, msg) \
    do { \
        if (cnd_value) { \
            if (errno) \
                perror(msg); \
            else \
                fprintf(stderr, "error: %s\n", msg); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define CSC(call) \
    do { \
        hipError_t res = call; \
        if (res != hipSuccess) { \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n", \
                __FILE__, __LINE__, hipGetErrorString(res)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

__host__ __device__ inline int mapping(
    const uint x, const uint y, const uint w, const uint h
) {
    return (y % h) * w + (x % w);
}

__global__ static void transpose(
    const dbl * const __restrict__ input, dbl * const __restrict__ output,
    const uint w, const uint h
) {
    __shared__ dbl title[TITLE_DIM][TITLE_DIM];

    const uint
        idxX = blockDim.x * blockIdx.x, idxY = blockDim.y * blockIdx.y,
        offsetX = blockDim.x * gridDim.x, offsetY = blockDim.y * gridDim.y;
    for (uint y = idxY; y < h; y += offsetY) {
        for (uint x = idxX; x < w; x += offsetX) {
            uint i = x + threadIdx.x, j = y + threadIdx.y;
            title[threadIdx.x][threadIdx.y] = input[mapping(i, j, w, h)];

            __syncthreads();

            i = x + threadIdx.y;
            j = y + threadIdx.x;
            if (i < w && j < h)
                output[mapping(j, i, h, w)] = title[threadIdx.y][threadIdx.x];
            __syncthreads();
        }
    }
}

static ostream &outMatrix(
    ostream &os,
    const dbl * const __restrict__ matrix,
    const uint w, const uint h
) {
    for (uint j = 0; j < h; ++j) {
        for (uint i = 0; i < w; ++i)
            os << setprecision(10) << scientific << matrix[j * w + i] << ' ';
        os << endl;
    }
    return os;
}

int main(const int argc, char ** const argv) {
    ios_base::sync_with_stdio(false);
    cerr.tie(nullptr);
    cin.tie(nullptr);

    uint w, h;
    cin >> h >> w;
    const uint size = sizeof(dbl) * w * h;
    dbl * const __restrict__ hostMatrix = (dbl *) malloc(size);
    exit_if(hostMatrix == NULL, "malloc()");

    for (uint j = 0; j < h; ++j)
        for (uint i = 0; i < w; ++i)
            cin >> hostMatrix[j * w + i];

    dbl *deviceInput, *deviceOutput;
    CSC(hipMalloc(&deviceInput, size));
    CSC(hipMemcpy(deviceInput, hostMatrix, size, hipMemcpyHostToDevice));
    CSC(hipMalloc(&deviceOutput, size));

    transpose<<<dim3(TITLE_DIM, TITLE_DIM), dim3(TITLE_DIM, TITLE_DIM)>>>(
        deviceInput, deviceOutput, w, h
    );
    CSC(hipGetLastError());

    CSC(hipMemcpy(hostMatrix, deviceOutput, size, hipMemcpyDeviceToHost));
    CSC(hipFree(deviceInput));
    CSC(hipFree(deviceOutput));

    outMatrix(cout, hostMatrix, w, h);
    free(hostMatrix);

    return 0;
}
