
#include <hip/hip_runtime.h>
#include <errno.h>
#include <limits.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

typedef unsigned uint;
typedef float flt;

#define exit_if(cnd_value, msg) \
    do { \
        if (cnd_value) { \
            if (errno) \
                perror(msg); \
            else \
                fprintf(stderr, "error: %s\n", msg); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define cudaErrorCheck(error) \
    do { \
        hipError_t res = error; \
        if (res != hipSuccess) { \
            fprintf(stderr, "cuda %s:%d error: %s\n", __FILE__, __LINE__, \
                hipGetErrorString(res)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

#define MU_COUNT (3U)
__device__ __constant__ int3 deviceMu[MU_COUNT];

__global__ static void classify(uchar4 * __restrict__, uint, uint);

__host__ static uchar4 *read_file(const char *, uint *, uint *);
__host__ static void write_file(
    const char *, const uchar4 * __restrict__, uint, uint
);

int main(void) {
    char fileIn[FILENAME_MAX], fileOut[FILENAME_MAX];
    scanf("%s%s", fileIn, fileOut);

    uint w, h;
    uchar4 * const __restrict__ data = read_file(fileIn, &w, &h);
    const uint size = sizeof(uchar4) * w * h;

    uchar4 *deviceData;
    cudaErrorCheck(hipMalloc(&deviceData, size));
    cudaErrorCheck(hipMemcpy(deviceData, data, size, hipMemcpyHostToDevice));

    int3 hostMu[MU_COUNT] = {
        make_int3(255, 0, 0),
        make_int3(0, 255, 0),
        make_int3(0, 0, 255)
    };
    cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(deviceMu), hostMu,
        sizeof(int3) * MU_COUNT
    ));

    classify<<<dim3(16U, 16U), dim3(16U, 16U)>>>(deviceData, w, h);
    cudaErrorCheck(hipGetLastError());
    cudaErrorCheck(hipMemcpy(data, deviceData, size, hipMemcpyDeviceToHost));
    cudaErrorCheck(hipFree(deviceData));

    write_file(fileOut, data, w, h);
    free(data);

    return 0;
}

__global__ static void classify(
    uchar4 * const __restrict__ data,
    const uint w, const uint h
) {
    const uint
        idxX = threadIdx.x + blockDim.x * blockIdx.x,
        idxY = threadIdx.y + blockDim.y * blockIdx.y,
        offsetX = blockDim.x * gridDim.x,
        offsetY = blockDim.y * gridDim.y;
    for (uint j = idxY; j < h; j += offsetY) {
        for (uint i = idxX; i < w; i += offsetX) {
            int min_distance = INT_MAX;
            const uint index = j * w + i;
            const uchar4 data_i_j = data[index];
            for (uint idx = 0; idx < MU_COUNT; ++idx) {
                const int distance =
                    (data_i_j.x - deviceMu[idx].x) * (data_i_j.x - deviceMu[idx].x) +
                    (data_i_j.y - deviceMu[idx].y) * (data_i_j.y - deviceMu[idx].y) +
                    (data_i_j.z - deviceMu[idx].z) * (data_i_j.z - deviceMu[idx].z);

                if (distance < min_distance) {
                    data[index].w = idx;
                    min_distance = distance;
                }
            }
        }
    }
}

__host__ static uchar4 *read_file(
    const char * const file, uint * const w, uint * const h
) {
    FILE * const stream = fopen(file, "rb");
    exit_if(stream == NULL, "fopen()");

    exit_if(fread(w, sizeof(uint), 1, stream) != 1, "fread()");
    exit_if(fread(h, sizeof(uint), 1, stream) != 1, "fread()");

    const uint count = (*w) * (*h);
    uchar4 * const data = (uchar4 *) malloc(sizeof(uchar4) * count);
    exit_if(data == NULL, "malloc()");

    exit_if(fread(data, sizeof(uchar4), count, stream) != count, "fread()");
    exit_if(fclose(stream) != 0, "close()");

    return data;
}

__host__ static void write_file(
    const char * const file, const uchar4 * const __restrict__ data,
    const uint w, const uint h
) {
    FILE * const stream = fopen(file, "wb");
    exit_if(stream == NULL, "fopen()");

    const uint count = w * h;
    exit_if(fwrite(&w, sizeof(uint), 1, stream) != 1, "fwrite()");
    exit_if(fwrite(&h, sizeof(uint), 1, stream) != 1, "fwrite()");
    exit_if(fwrite(data, sizeof(uchar4), count, stream) != count, "fwrite()");
    exit_if(fclose(stream) != 0, "fclose()");
}
