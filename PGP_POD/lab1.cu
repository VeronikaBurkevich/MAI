
#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>

using namespace std;

void Error(hipError_t error)
{
    if (error != hipSuccess){ 
        cout << "ERROR:" << hipGetErrorString(error) << endl; 
        exit(0); 
    }
}
__global__ void sqr_items_vectors(double* a, double* result, int n)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    while (tid < n) {
        result[tid] = a[tid]*a[tid];
        tid += blockDim.x*gridDim.x;
    }
}

int main()
{
    int n, size;
    double *a, *result;
    double *gpu_a, *gpu_result;

    cin >> n;

    size = sizeof(double) * n;

    a = (double*)malloc(size);
    result = (double*)malloc(size);

    for (int i = 0; i < n; ++i) {
        cin >> a[i];
    }

    Error(hipMalloc(&gpu_a, size));
    Error(hipMalloc(&gpu_result, size));
    Error(hipMemcpy(gpu_a, a, size, hipMemcpyHostToDevice));

    sqr_items_vectors<<<256, 256>>>(gpu_a, gpu_result, n);

    Error(hipMemcpy(result, gpu_result, size, hipMemcpyDeviceToHost));

    for (int i = 0; i < n; ++i) {
        cout << scientific << setprecision(10) << result[i] << " ";
    }
    cout << endl;

    Error(hipFree(gpu_a));
    Error(hipFree(gpu_result));
    free(a);
    free(result);

    return 0;
}
