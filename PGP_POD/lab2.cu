#include "hip/hip_runtime.h"
#include <errno.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

typedef unsigned char uchar;
typedef unsigned uint;
typedef double dbl;

#define exit_if(cnd_value, msg) \
    do { \
        if (cnd_value) \
        { \
            if (errno) \
                perror(msg); \
            else \
                fprintf(stderr, "error: %s\n", msg); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define cudaErrorCheck(error) \
    do { \
        hipError_t result = error; \
        if (result != hipSuccess) { \
            fprintf(stderr, "cuda %s:%d error: %s\n", __FILE__, __LINE__, \
                hipGetErrorString(result)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

texture<uchar4, hipTextureType2D, hipReadModeElementType> tex;

__device__ __host__ int my_min(const int a, const int b) {
    const int diff = a - b, shift = CHAR_BIT * sizeof(int) - 1;
    return b + (diff & (diff >> shift));
}

__device__ dbl filter(const uchar4 pixel) {
    return pixel.x * 0.299 + pixel.y * 0.587 + pixel.z * 0.114;
}

__global__ void kernel(
    uchar4 * const __restrict__ data,
    const uint w, const uint h
) {
    const uint
        idxX = threadIdx.x + blockDim.x * blockIdx.x,
        idxY = threadIdx.y + blockDim.y * blockIdx.y,
        offsetX = blockDim.x * gridDim.x, offsetY = blockDim.y * gridDim.y;
    for (uint x = idxX; x < w; x += offsetX)
        for (uint y = idxY; y < h; y += offsetY) {
            const uchar4 pixel = tex2D(tex, x, y);
            const uchar value = (uchar) my_min((int) filter(pixel), UCHAR_MAX);
            data[y * w + x] = make_uchar4(value, value, value, pixel.w);
        }
}

int main() {
    char fileIn[FILENAME_MAX], fileOut[FILENAME_MAX];
    scanf("%s%s", fileIn, fileOut);

    FILE * const streamIn = fopen(fileIn, "rb");
    exit_if(!streamIn, "fopen()");

    uint w, h;
    fread(&w, sizeof(uint), 1, streamIn);
    fread(&h, sizeof(uint), 1, streamIn);

    uchar4 * const img = (uchar4 *) malloc(sizeof(uchar4) * w * h);
    exit_if(img == NULL, "malloc()");

    fread(img, sizeof(uchar4), h * w, streamIn);
    fclose(streamIn);

    hipChannelFormatDesc channel = hipCreateChannelDesc<uchar4>();
    cudaErrorCheck(hipGetLastError());

    hipArray *deviceArray;
    cudaErrorCheck(hipMallocArray(&deviceArray, &channel, w, h));
    cudaErrorCheck(hipMemcpyToArray(deviceArray, 0, 0, img,
        sizeof(uchar4) * w * h, hipMemcpyHostToDevice));

    tex.addressMode[0] = hipAddressModeClamp;
    tex.addressMode[1] = hipAddressModeClamp;
    tex.channelDesc = channel;
    tex.filterMode = hipFilterModePoint;
    tex.normalized = false;
    cudaErrorCheck(hipBindTextureToArray(tex, deviceArray, channel));

    uchar4 *deviceImg;
    cudaErrorCheck(hipMalloc(&deviceImg, sizeof(uchar4) * w * h));
    kernel<<<dim3(16U, 16U), dim3(16U, 16U)>>>(deviceImg, w, h);
    cudaErrorCheck(hipGetLastError());
    cudaErrorCheck(hipMemcpy(img, deviceImg, sizeof(uchar4) * w * h,
        hipMemcpyDeviceToHost));

    FILE * const streamOut = fopen(fileOut, "wb");
    exit_if(streamOut == NULL, "fopen()");

    fwrite(&w, sizeof(uint), 1, streamOut);
    fwrite(&h, sizeof(uint), 1, streamOut);
    fwrite(img, sizeof(uchar4), w * h, streamOut);
    fclose(streamOut);

    cudaErrorCheck(hipUnbindTexture(tex));
    cudaErrorCheck(hipFreeArray(deviceArray));
    cudaErrorCheck(hipFree(deviceImg));
    free(img);

    return 0;
}
