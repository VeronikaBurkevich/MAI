#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>

#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>

#define MAX_VALUE ((1UL << 24) + 1U)
#define BLOCK_DIM (16U)
#define  GRID_DIM (16U)

typedef unsigned uint;

#define CSC(call) \
    do { \
        hipError_t res = call; \
        if (res != hipSuccess) { \
            fprintf(stderr, "ERROR in %s:%d. Message: %s\n", \
                __FILE__, __LINE__, hipGetErrorString(res)); \
            exit(0); \
        } \
    } while(0)

__global__ static void hist(
    const int * __restrict__ const input,
    int * __restrict__ const counts,
    const uint count
) {
    const uint offset = blockDim.x * gridDim.x;
    uint idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (; idx < count; idx += offset)
	   atomicAdd(counts + input[idx], 1);
}

__global__ static void sort(
    const int * __restrict__ const input,
    int * __restrict__ const counts,
    int * __restrict__ const output,
    const uint count
) {
    const uint offset = blockDim.x * gridDim.x;
    uint idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (; idx < count; idx += offset) {
        const int i = atomicAdd(counts + input[idx], -1) - 1;
        output[i] = input[idx];
    }
}

int main(void) {
    uint count;
    fread(&count, sizeof(uint), 1U, stdin);
    const uint size = sizeof(int) * count;

    int *buffer = (int *) malloc(size);
    fread(buffer, sizeof(int), count, stdin);

    int *deviceInput;
    CSC(hipMalloc(&deviceInput, size));
    CSC(hipMemcpy(deviceInput, buffer, size, hipMemcpyHostToDevice));

    int *deviceCounts;
    CSC(hipMalloc(&deviceCounts, sizeof(int) * MAX_VALUE));
    CSC(hipMemset(deviceCounts, 0, sizeof(int) * MAX_VALUE));

    int *deviceOutput;
    CSC(hipMalloc(&deviceOutput, size));

    hipEvent_t startTime, endTime;
    CSC(hipEventCreate(&startTime));
    CSC(hipEventCreate(&endTime));
    CSC(hipEventRecord(startTime));

    hist<<<GRID_DIM, BLOCK_DIM>>>(deviceInput, deviceCounts, count);
    thrust::inclusive_scan(thrust::device,
        deviceCounts, deviceCounts + MAX_VALUE, deviceCounts
    );
    sort<<<GRID_DIM, BLOCK_DIM>>>(deviceInput, deviceCounts, deviceOutput, count);
    CSC(hipGetLastError());

    CSC(hipEventRecord(endTime));
    CSC(hipEventSynchronize(endTime));
    float t;
    CSC(hipEventElapsedTime(&t, startTime, endTime));
    CSC(hipEventDestroy(startTime));
    CSC(hipEventDestroy(endTime));

    CSC(hipMemcpy(buffer, deviceOutput, sizeof(int) * count,
        hipMemcpyDeviceToHost
    ));
    CSC(hipFree(deviceInput));
    CSC(hipFree(deviceOutput));

    fwrite(buffer, sizeof(int), count, stdout);
    free(buffer);

    return 0;
}
