#include <errno.h>
#include <float.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <string>
#include <vector>

#include "hip/hip_runtime.h"


typedef unsigned char uchar;
typedef unsigned uint;
typedef float flt;

using namespace std;

#define FLT_1_PHI      (0.61803398874989485F)
#define FLT_PHI        (1.61803398874989484F)
#define FLT_PI         (3.14159265358979323F)
#define FLT_SQRT3      (1.73205080756887729F)
#define FLT_SQRT_PHI_2 (1.90211303259030714F)

#define exit_if(cnd_value, msg) \
    do { \
        if ((cnd_value)) { \
            if (errno) \
                perror((msg)); \
            else \
                fprintf(stderr, "error: %s\n", (msg)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

#define cudaErrorCheck(error) \
    do { \
        hipError_t result = (error); \
        if (res != hipSuccess) { \
            fprintf(stderr, "cuda %s:%d error: %s\n", __FILE__, __LINE__, \
                hipGetErrorString(res)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

struct triangle {
    float3 a, b, c;
    uchar4 color;
};

static const uchar4
    // aqua         = {   0, 255, 255, 0 },
    darkgreen    = {   0, 100,   0, 0 },
    darkred      = { 139,   0,   0, 0 },
    gold         = { 255, 215,   0, 0 },
    green        = {   0, 255,   0, 0 },
    // khaki        = { 240, 230, 140, 0 },
    // magenta      = { 255,   0, 255, 0 },
    maroon       = { 128,   0,   0, 0 },
    midnightblue = {  25,  25, 112, 0 },
    purple       = { 128,   0, 128, 0 },
    red          = { 255,   0,   0, 0 },
    teal         = {   0, 128, 128, 0 },
    yellow       = { 255, 255,   0, 0 };

__device__ __host__ static float3 cross(const float3 a, const float3 b) {
    return {
        a.y * b.z - a.z * b.y,
        a.z * b.x - a.x * b.z,
        a.x * b.y - a.y * b.x
    };
}

__device__ __host__ static flt dot(const float3 a, const float3 b) {
    return a.x * b.x + a.y * b.y + a.z * b.z;
}

__device__ __host__ static float3 float3_multiplies(
    const float3 a,
    const flt alpha
) {
    return { a.x * alpha, a.y * alpha, a.z * alpha };
}

__device__ __host__ static float3 float3_plus(
    const float3 a,
    const float3 b
) {
    return { a.x + b.x, a.y + b.y, a.z + b.z };
}

__device__ __host__ static float3 float3_fma(
    const float3 a,
    const flt radius,
    const float3 center
) {
    return float3_plus(float3_multiplies(a, radius), center);
}

__device__ __host__ static flt length(const float3 a) {
    return sqrtf(dot(a, a));
}

__device__ __host__ static float3 float3_minus(const float3 a, const float3 b) {
    return { a.x - b.x, a.y - b.y, a.z - b.z };
}

__device__ __host__ static float3 float3_multiplies(
    const float3 a,
    const float3 b
) {
    return { a.x * b.x, a.y * b.y, a.z * b.z };
}

__device__ __host__ static float3 normalize(const float3 a) {
    const flt a_length = length(a);
    return { a.x / a_length, a.y / a_length, a.z / a_length };
}

__device__ __host__ float3 mult(
    const float3 a,
    const float3 b,
    const float3 c,
    const float3 all
) {
    return {
        a.x * all.x + b.x * all.y + c.x * all.z,
        a.y * all.x + b.y * all.y + c.y * all.z,
        a.z * all.x + b.z * all.y + c.z * all.z
    };
}

static void make_dodecahedron(
    vector<triangle> &triangles,
    const flt radius,
    const float3 center
) {
    static const float3
        point_1  = { -FLT_1_PHI / FLT_SQRT3, 0.0F, FLT_PHI / FLT_SQRT3 },
        point_2  = {  FLT_1_PHI / FLT_SQRT3, 0.0F, FLT_PHI / FLT_SQRT3 },
        point_3  = { -1.0F / FLT_SQRT3, 1.0F / FLT_SQRT3, 1.0F / FLT_SQRT3 },
        point_4  = { 1.0F / FLT_SQRT3, 1.0F / FLT_SQRT3, 1.0F / FLT_SQRT3 },
        point_5  = { 1.0F / FLT_SQRT3, -1.0F / FLT_SQRT3, 1.0F / FLT_SQRT3 },
        point_6  = { -1.0F / FLT_SQRT3, -1.0F / FLT_SQRT3, 1.0F / FLT_SQRT3 },
        point_7  = { 0.0F, -FLT_PHI / FLT_SQRT3, FLT_1_PHI / FLT_SQRT3 },
        point_8  = { 0.0F, FLT_PHI / FLT_SQRT3, FLT_1_PHI / FLT_SQRT3 },
        point_9  = { -FLT_PHI / FLT_SQRT3, -FLT_1_PHI / FLT_SQRT3, 0.0F },
        point_10 = { -FLT_PHI / FLT_SQRT3, FLT_1_PHI / FLT_SQRT3, 0.0F },
        point_11 = { FLT_PHI / FLT_SQRT3, FLT_1_PHI / FLT_SQRT3, 0.0F },
        point_12 = { FLT_PHI / FLT_SQRT3, -FLT_1_PHI / FLT_SQRT3, 0.0F },
        point_13 = { 0.0F, -FLT_PHI / FLT_SQRT3, -FLT_1_PHI / FLT_SQRT3 },
        point_14 = { 0.0F, FLT_PHI / FLT_SQRT3, -FLT_1_PHI / FLT_SQRT3 },
        point_15 = { 1.0F / FLT_SQRT3, 1.0F / FLT_SQRT3, -1.0F / FLT_SQRT3 },
        point_16 = { 1.0F / FLT_SQRT3, -1.0F / FLT_SQRT3, -1.0F / FLT_SQRT3 },
        point_17 = { -1.0F / FLT_SQRT3, -1.0F / FLT_SQRT3, -1.0F / FLT_SQRT3 },
        point_18 = { -1.0F / FLT_SQRT3, 1.0F / FLT_SQRT3, -1.0F / FLT_SQRT3 },
        point_19 = {  FLT_1_PHI / FLT_SQRT3, 0.0F, -FLT_PHI / FLT_SQRT3 },
        point_20 = { -FLT_1_PHI / FLT_SQRT3 , 0.0F, -FLT_PHI / FLT_SQRT3 };

    triangles.push_back({ point_5, point_1, point_7, green });
    triangles.push_back({ point_1, point_6, point_7, darkgreen });
    triangles.push_back({ point_1, point_5, point_2, maroon });
    triangles.push_back({ point_1, point_4, point_8, purple });
    triangles.push_back({ point_3, point_1, point_8, red });
    triangles.push_back({ point_1, point_2, point_4, yellow });
    triangles.push_back({ point_11, point_2, point_12, green });
    triangles.push_back({ point_4, point_2, point_11, darkgreen });
    triangles.push_back({ point_2, point_5, point_12, maroon });
    triangles.push_back({ point_6, point_1, point_9, purple });
    triangles.push_back({ point_1, point_3, point_10, red });
    triangles.push_back({ point_9, point_1, point_10, yellow });
    triangles.push_back({ point_6, point_9, point_17, green });
    triangles.push_back({ point_7, point_6, point_13, darkgreen });
    triangles.push_back({ point_13, point_6, point_17, maroon });
    triangles.push_back({ point_5, point_13, point_16, purple });
    triangles.push_back({ point_5, point_7, point_13, red });
    triangles.push_back({ point_12, point_5, point_16, green });
    triangles.push_back({ point_3, point_14, point_18, darkgreen });
    triangles.push_back({ point_3, point_8, point_14, maroon });
    triangles.push_back({ point_10, point_3, point_18, purple });
    triangles.push_back({ point_14, point_4, point_15, red });
    triangles.push_back({ point_8, point_4, point_14, yellow });
    triangles.push_back({ point_4, point_11, point_15, green });
    triangles.push_back({ point_9, point_18, point_20, darkgreen });
    triangles.push_back({ point_17, point_9, point_20, maroon });
    triangles.push_back({ point_9, point_10, point_18, purple });
    triangles.push_back({ point_15, point_12, point_19, red });
    triangles.push_back({ point_12, point_16, point_19, green });
    triangles.push_back({ point_11, point_12, point_15, darkgreen });
    triangles.push_back({ point_13, point_20, point_19, maroon });
    triangles.push_back({ point_16, point_13, point_19, purple });
    triangles.push_back({ point_13, point_17, point_20, yellow });
    triangles.push_back({ point_20, point_14, point_19, maroon });
    triangles.push_back({ point_18, point_14, point_20, darkgreen });
    triangles.push_back({ point_14, point_15, point_19, yellow });

    for (size_t i = 0; i < 36U; ++i) {
        triangle &triangle_ref = triangles[triangles.size() - i - 1U];
        triangle_ref.a = float3_fma(triangle_ref.a, radius, center);
        triangle_ref.b = float3_fma(triangle_ref.b, radius, center);
        triangle_ref.c = float3_fma(triangle_ref.c, radius, center);
    }
}

static void make_icosagedron(
    vector<triangle> &triangles,
    const flt radius,
    const float3 center
) {
    static const float3
        point_1  = { 0.0F, -1.0F / FLT_SQRT_PHI_2, FLT_PHI / FLT_SQRT_PHI_2 },
        point_2  = { 0.0F,  1.0 / FLT_SQRT_PHI_2,  FLT_PHI / FLT_SQRT_PHI_2 },
        point_3  = { -FLT_PHI / FLT_SQRT_PHI_2, 0.0, 1.0 / FLT_SQRT_PHI_2 },
        point_4  = { FLT_PHI / FLT_SQRT_PHI_2, 0.0, 1.0 / FLT_SQRT_PHI_2 },
        point_5  = { -1.0 / FLT_SQRT_PHI_2, FLT_PHI / FLT_SQRT_PHI_2, 0.0 },
        point_6  = { 1.0 / FLT_SQRT_PHI_2, FLT_PHI / FLT_SQRT_PHI_2, 0.0 },
        point_7  = { 1.0 / FLT_SQRT_PHI_2, -FLT_PHI / FLT_SQRT_PHI_2, 0.0 },
        point_8  = { -1.0 / FLT_SQRT_PHI_2, -FLT_PHI / FLT_SQRT_PHI_2, 0.0 },
        point_9  = { -FLT_PHI / FLT_SQRT_PHI_2, 0.0, -1.0 / FLT_SQRT_PHI_2 },
        point_10 = { FLT_PHI / FLT_SQRT_PHI_2, 0.0, -1.0 / FLT_SQRT_PHI_2 },
        point_11 = { 0.0, -1.0 / FLT_SQRT_PHI_2, -FLT_PHI / FLT_SQRT_PHI_2 },
        point_12 = { 0.0, 1.0 / FLT_SQRT_PHI_2, -FLT_PHI / FLT_SQRT_PHI_2 };

    triangles.push_back({ point_1, point_2, point_3, purple });
    triangles.push_back({ point_2, point_1, point_4, red });
    triangles.push_back({ point_1, point_3, point_8, yellow });
    triangles.push_back({ point_3, point_2, point_5, green });
    triangles.push_back({ point_5, point_2, point_6, darkgreen });
    triangles.push_back({ point_7, point_1, point_8, maroon });
    triangles.push_back({ point_4, point_1, point_7, purple });
    triangles.push_back({ point_2, point_4, point_6, red });
    triangles.push_back({ point_5, point_6, point_12, yellow });
    triangles.push_back({ point_7, point_8, point_11, green });
    triangles.push_back({ point_4, point_7, point_10, darkgreen });
    triangles.push_back({ point_6, point_4, point_10, maroon });
    triangles.push_back({ point_8, point_3, point_9, purple });
    triangles.push_back({ point_3, point_5, point_9, red });
    triangles.push_back({ point_10, point_11, point_12, yellow });
    triangles.push_back({ point_11, point_9, point_12, green });
    triangles.push_back({ point_6, point_10, point_12, darkgreen });
    triangles.push_back({ point_10, point_7, point_11, maroon });
    triangles.push_back({ point_8, point_9, point_11, purple });
    triangles.push_back({ point_9, point_5, point_12, red });

    for (size_t i = 0; i < 20U; ++i) {
        triangle &triangle_ref = triangles[triangles.size() - i - 1U];
        triangle_ref.a = float3_fma(triangle_ref.a, radius, center);
        triangle_ref.b = float3_fma(triangle_ref.b, radius, center);
        triangle_ref.c = float3_fma(triangle_ref.c, radius, center);
    }
}

static void make_tetrahedron(
    vector<triangle> &triangles,
    const flt radius,
    const float3 center
) {
    static const float3
        point_1 = { sqrtf(8.0F / 9.0F), 0.0F, -1.0F / 3.0F },
        point_2 = { -sqrtf(2.0F / 9.0F), sqrtf(2.0F / 3.0F), -1.0F / 3.0F },
        point_3 = { -sqrtf(2.0F / 9.0F), -sqrtf(2.0F / 3.0F), -1.0F / 3.0F },
        point_4 = { 0.0F, 0.0F, 1.0F };

    triangles.push_back({ point_2, point_1, point_3, gold });
    triangles.push_back({ point_1, point_2, point_4, darkred });
    triangles.push_back({ point_3, point_1, point_4, teal });
    triangles.push_back({ point_2, point_3, point_4, midnightblue });

    for (size_t i = 0; i < 4U; ++i) {
        triangle &triangle_ref = triangles[triangles.size() - i - 1U];
        triangle_ref.a = float3_fma(triangle_ref.a, radius, center);
        triangle_ref.b = float3_fma(triangle_ref.b, radius, center);
        triangle_ref.c = float3_fma(triangle_ref.c, radius, center);
    }
}

static void create_field(
    vector<triangle> &triangles,
    const float3 point_a,
    const float3 point_b,
    const float3 point_c,
    const float3 point_d,
    const float3 texture_color
) {
    const uchar4 color = {
        (uchar) (texture_color.x * UCHAR_MAX),
        (uchar) (texture_color.y * UCHAR_MAX),
        (uchar) (texture_color.z * UCHAR_MAX),
        0
    };

    triangles.push_back({ point_a, point_b, point_c, color });
    triangles.push_back({ point_a, point_c, point_d, color });
}

__device__ __host__ static uchar4 ray(
    float3 pos,
    float3 dir,
    const float3 sunshine,
    const float3 sunshine_c,
    triangle * const triangles,
    const size_t count
) {
    ptrdiff_t min_value = -1;
    flt ts_min;
    for (size_t i = 0; i < count; ++i) {
        const float3
            e1 = float3_minus(triangles[i].b, triangles[i].a),
            e2 = float3_minus(triangles[i].c, triangles[i].a),
            p = cross(dir, e2);
        const flt div = dot(p, e1);
        if (fabsf(div) < FLT_EPSILON)
            continue;
        const float3 t = float3_minus(pos, triangles[i].a);
        const flt u = dot(p, t) / div;
        if (u < 0.0F || u > 1.0F)
            continue;
        float3 q = cross(t, e1);
        flt v = dot(q, dir) / div;
        if (v < 0.0F || v + u > 1.0F)
            continue;
        flt ts = dot(q, e2) / div;
        if (ts < 0.0F)
            continue;
        if (min_value == -1 || ts < ts_min) {
            min_value = i;
            ts_min = ts;
        }
    }

    if (min_value == -1)
        return { 0, 0, 0, 0 };

    pos = float3_plus(float3_multiplies(dir, ts_min), pos);
    dir = float3_minus(sunshine, pos);
    const flt dir_length = length(dir);
    dir = normalize(dir);

    for (size_t i = 0; i < count; ++i) {
        const float3
            e1 = float3_minus(triangles[i].b, triangles[i].a),
            e2 = float3_minus(triangles[i].c, triangles[i].a),
            p = cross(dir, e2);
        const flt div = dot(p, e1);

        if (fabsf(div) < FLT_EPSILON)
            continue;
        float3 t = float3_minus(pos, triangles[i].a);
        const flt u = dot(p, t) / div;
        if (u < 0.0F || u > 1.0F)
            continue;
        float3 q = cross(t, e1);
        const flt v = dot(q, dir) / div;

        if (v < 0.0F || v + u > 1.0F)
            continue;

        const flt ts = dot(q, e2) / div;

        if (ts > 0.0F && ts < dir_length && i != min_value)
            return { 0, 0, 0, 0 };
    }

    const uchar4 color_min = triangles[min_value].color;
    const float3 result = float3_multiplies(
        make_float3(color_min.x, color_min.y, color_min.z),
        sunshine_c
    );
    return make_uchar4(result.x, result.y, result.z, color_min.w);
}

static void render_cpu(
    float3 pc, float3 pv,
    const uint w, const uint h,
    flt angle,
    uchar4 * const data,
    float3 sunshine, float3 sunshine_c,
    triangle * const triangles, const size_t count
) {
    const flt dw = 2.0F / (w - 1), dh = 2.0F / (h - 1),
        z = 1.0F / tanf(angle * FLT_PI / 360.0F);
    const float3
        bz = normalize(float3_minus(pv, pc)),
        bx = normalize(cross(bz, { 0.0F, 0.0F, 1.0F })),
        by = normalize(cross(bx, bz));
    for (uint i = 0; i < w; ++i)
        for (uint j = 0; j < h; ++j) {
            const float3 v = { -1.0F + dw * i, (-1.0F + dh * j) * h / w, z };
            const float3 dir = mult(bx, by, bz, v);
            data[(h - 1 - j) * w + i] = ray(pc, normalize(dir), sunshine, sunshine_c, triangles, count);
        }
}

__global__ static void render(
    const float3 pc, const float3 pv,
    const uint w, const uint h,
    const flt angle,
    uchar4 * const data,
    const float3 sunshine, const float3 sunshine_c,
    triangle * const triangles, const size_t count
) {
    const uint
        idx = blockDim.x * blockIdx.x + threadIdx.x,
        idy = blockDim.y * blockIdx.y + threadIdx.y,
        offsetx = blockDim.x * gridDim.x,
        offsety = blockDim.y * gridDim.y;

    const flt dw = 2.0F / (w - 1), dh = 2.0F / (h - 1),
        z = 1.0F / tanf(angle * FLT_PI / 360.0F);
    const float3
        bz = normalize(float3_minus(pv, pc)),
        bx = normalize(cross(bz, { 0.0F, 0.0F, 1.0F })),
        by = normalize(cross(bx, bz));
    for (uint i = idx; i < w; i += offsetx)
        for (uint j = idy; j < h; j += offsety) {
            const float3 v = { -1.0F + dw * i, (-1.0F + dh * j) * h / w, z },
                dir = mult(bx, by, bz, v);
            data[(h - 1 - j) * w + i] = ray(pc, normalize(dir), sunshine, sunshine_c, triangles, count);
        }
}

int main(const int argc, char ** const argv) {
    int flag_cuda, numbers_data, width, height, iter, light;
    float three_points_1, three_points_2, phi, r_first_0, z_first_0, f_first_0, A_first_r, A_first_z, w_first_r,
    w_first_z, w_first_f, p_first_r, p_first_z, r_second_0, z_second_0, f_second_0, A_second_r, A_second_z, w_second_r,
    w_second_z, w_second_f, p_second_r, p_second_z, rds, r_, trs, lght, tro;
    float3 cntr, clr, tp_1, tp_2, tp_3, tp_4, tc, pc, pv, sunshine, sunshine_c;
    string trash;
    char bffr[256];

    string exit_place;
    if (argc >= 3) {
        cout << "SOMETHING WENT WRONG......\nYOU CAN USE EQUAL OR LESS THEN 2 ARGUMENTS!\n";
        return 0;
    }
    if (argc == 1)
        flag_cuda = 1;
    else if (!strcmp(argv[1], "--gpu"))
        flag_cuda = 1;
    else if (!strcmp(argv[1], "--default")) {
        cout << "99"
            "./images_data/"
            "640 480 120\n"
            "6.8 2.7 0.1 2.2 1.1 2.1 6.1 1.1 0.0 0.0\n"
            "2.2 0.001 0.01 0.54 0.12 1.1 4.2 1.1 0.0 0.0\n"
            "3.0 0.0 0.0 255 255 255 1.0 42 42 42\n"
            "0.0 3.0 0.0 255 255 255 0.75 42 42 42\n"
            "0.0 0.0 0.0 255 255 255 1.0 42 42 42\n"
            "-5.0 -5.0 -1.0 -5.0 5.0 -1.0 5.0 5.0 -1.0 5.0 -5.0 -1.0 ~/textures 0.0 1.0 0.0 0.5\n"
            "1\n"
            "-8.0 0.0 8.0 1.0 1.0 1.0\n"
            "100 100\n";
        return 0;
    }
    else if (!strcmp(argv[1], "--cpu"))
        flag_cuda = 0;

    cin >> numbers_data >> exit_place >> width >> height >> phi;
    cin >> r_first_0 >> z_first_0 >> f_first_0 >> A_first_r >> A_first_z >> w_first_r >> w_first_z >> w_first_f >> p_first_r >> p_first_z >>
    r_second_0 >> z_second_0 >> f_second_0 >> A_second_r >> A_second_z >> w_second_r >> w_second_z >> w_second_f >> p_second_r >> p_second_z;

    uchar4 *data = (uchar4 *) malloc(width * height * sizeof(uchar4));
    uchar4 *deviceData;
    hipMalloc(&deviceData, width * height * sizeof(uchar4));
    hipMemset(deviceData, 0, width * height * sizeof(uchar4));

    vector <triangle> triangles;
    cin >> cntr.x >> cntr.y >> cntr.z >> clr.x >> clr.y >> clr.z >> rds >> r_ >> trs >> lght;
    make_tetrahedron(triangles, rds, cntr);

    cin >> cntr.x >> cntr.y >> cntr.z >> clr.x >> clr.y >> clr.z >> rds >> r_ >> trs >> lght;
    make_dodecahedron(triangles, rds, cntr);

    cin >> cntr.x >> cntr.y >> cntr.z >> clr.x >> clr.y >> clr.z >> rds >> r_ >> trs >> lght;
    make_icosagedron(triangles, rds, cntr);
    cin >> tp_1.x >> tp_1.y >> tp_1.z >> tp_2.x >> tp_2.y >> tp_2.z >> tp_3.x >> tp_3.y >> tp_3.z >> tp_4.x >> tp_4.y >> tp_4.z;
    cin >> trash >> tc.x >> tc.y >> tc.z >> tro;

    create_field(triangles, tp_1, tp_2, tp_3, tp_4, tc);
    cin >> light;
    if (light != 1) {
        cout << "I need in only 3 points for this project!\n";
        return 0;
    }

    cin >> sunshine.x >> sunshine.y >> sunshine.z >> sunshine_c.x >> sunshine_c.y >> sunshine_c.z >> three_points_1 >> three_points_2;
    sunshine.x = 100;
    sunshine.y = 100;
    sunshine.z = 100;

    triangle *deviceTriangles;
    hipMalloc(&deviceTriangles, triangles.size() * sizeof(triangle));
    hipMemcpy(deviceTriangles, triangles.data(), triangles.size() * sizeof(triangle),
        hipMemcpyHostToDevice
    );

    iter = 0;
    while (iter < numbers_data) {
        float step = 2 * FLT_PI * iter / numbers_data;

        float r_first = A_first_r * sin(w_first_r * step + p_first_r) + r_first_0;
        float z_first = A_first_z * sin(w_first_z * step + p_first_z) + z_first_0;
        float f_first = w_first_f * step + f_first_0;

        float r_second = A_second_r * sin(w_second_r * step + p_second_r) + r_second_0;
        float z_second = A_second_z * sin(w_second_z * step + p_second_z) + z_second_0;
        float f_second = w_second_f * step + f_second_0;

        pc.x = cos(f_first) * r_first;
        pc.y = sin(f_first) * r_first;
        pc.z = z_first;

        pv.x = cos(f_second) * r_second;
        pv.y = sin(f_second) * r_second;
        pv.z = z_second;

        if (flag_cuda == 1) {
            render<<<dim3(16, 16), dim3(16, 16)>>>(pc, pv, width, height, phi, deviceData, sunshine, sunshine_c, deviceTriangles, triangles.size());
            hipMemcpy(data, deviceData, sizeof(uchar4) * width * height, hipMemcpyDeviceToHost);
        }
        else
            render_cpu(pc, pv, width, height, phi, data, sunshine, sunshine_c, triangles.data(), triangles.size());


        sprintf(bffr, "%d.data", iter);
        const string result_show(exit_place + "/" + string(bffr));

        cout << iter << ": " << bffr << endl;
        FILE* out = fopen(result_show.c_str(), "wb");
        fwrite(&width, sizeof(int), 1, out);
        fwrite(&height, sizeof(int), 1, out);
        fwrite(data, sizeof(uchar4), width * height, out);
        fclose(out);

        ++iter;
    }

    hipFree(deviceData);
    hipFree(deviceTriangles);
    free(data);

    return 0;
}
